
#include <hip/hip_runtime.h>
#include <stdio.h>

// declaração de uma constante (compartilhada c/ somente leitura c/ todas as threads)
__device__ const char *STR = "HELLO WORLD!";
const char STR_LENGTH = 12;

// GPU: Função imprime um letra por fluxo de execução.
__global__ void hello()
{
	printf("%c", STR[threadIdx.x % STR_LENGTH]);
}

// CPU: Função principal
int main (int argc, char ** argv) { 
	int nthreads = 12;
	int nblocos = 1;

	if (argc == 3) {
		nblocos = atoi(argv[1]);
		nthreads = atoi(argv[2]);
	} else {
                printf ("\n ############# \n");
		printf ("./02_hello <nblocos> <nthreads>\n");
		printf ("Caso não haja passagem de parâmetros, atribuiu-se:\n(1) nblocos c/ 1 e nthreads definido c/ 12\n");
	}

        // Função que será executada em GPU
        // Parâmetros do kernel (número de blocos e número de threads)
        // A função não recebe parâmetros
	hello<<<nblocos,nthreads>>>();

	// Sincronização das threads
	hipDeviceSynchronize();
	printf("\n");

	return 0;
}
