#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 32

// função executada na GPU
__global__ void vecAdd (int *Da, int *Db, int *Dc) {
   int i = threadIdx.x;
   Dc[i] = Da[i] + Db[i];
}

// função executada na CPU
__host__ void initvet(int *host_a, int *host_b) {
  // Inicialização dos vetores a e b
  for (int i=0; i < N; i++) {
    host_a[i] = N-i;
    host_b[i] = i;
  }
}

// função executada na CPU
__host__ void printvetores (int *a, int *b, int *c) {
  printf("\t [i] \t A\t B\t C\t \n");
  for (int i=0; i < N; i++)
    printf("\t [%d] \t %d\t %d\t %d\n", i, a[i], b[i], c[i]);
}

// função principal executada iniciada em CPU
int main(int argc, char const *argv[]) {
  int *a, *b, *c;
  int *dev_a, *dev_b, *dev_c;
  int size;

  size = sizeof(int)*N;
  
  // alocação de memória no HOST para os vetores (a,b e c)
  // a = (int *) malloc (size);
  b = (int *) malloc (size);
  // c = (int *) malloc (size);
  hipHostMalloc((void **) &a, size, hipHostMallocDefault);
  // cudaMallocHost((void **) &b, size);
  hipHostMalloc((void **) &c, size, hipHostMallocDefault);

  // Inicialização dos vetores
  initvet(a,b);

  // alocação de memória na GPU para os vetores (a,b e c)
  hipMalloc ((void **) &dev_a, size);
  hipMalloc ((void **) &dev_b, size);
  hipMalloc ((void **) &dev_c, size);

  // cópia dos vetores gerados em CPU p/ memória da GPU
  hipMemcpy (dev_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy (dev_b, b, size, hipMemcpyHostToDevice);

  // execução do kernel vecAdd em GPU.
  vecAdd<<<1,N>>>(dev_a, dev_b, dev_c);

  // cópia do vetor de resultado calculado em GPU p/ memória do HOST
  hipMemcpy (c, dev_c, size, hipMemcpyDeviceToHost);

  // impressão dos vetores
  printvetores (a, b, c);
  printf ("\n **** \n Nro Threads = %d\n Nro de Blocos = 1\n", N);

  // Libera memória da GPU
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  // Libera memória no HOST
  hipHostFree(a);
  hipHostFree(b);
  hipHostFree(c);

  return 0;
}
