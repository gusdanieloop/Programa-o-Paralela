#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N 2048
#define T 1024 // numero max de threads por bloco

// função executada na GPU
__global__ void vecAdd (int *da, int *db, int *dc) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < N) {
     dc[i] = da[i] + db[i];
   }
}

// função executada na CPU
__host__ void initvet(int *host_a, int *host_b) {
  // Inicialização dos vetores a e b
  for (int i=0; i < N; i++) {
    host_a[i] = N-i;
    host_b[i] = i;
  }
}

// função executada na CPU
__host__ void printvetores (int *a, int *b, int *c) {
  printf("\t [i] \t A\t B\t C\t \n");
  for (int i=0; i < N; i++) {
    if ((i % 100) == 0)
      printf("\t [%d] \t %d\t %d\t %d\n", i, a[i], b[i], c[i]);
  }
}

// função principal executada iniciada em CPU
int main(int argc, char const *argv[]) {
  int *a, *b, *c;
  int *dev_a, *dev_b, *dev_c;
  int size;

  size = N * sizeof(int);

  // Inicialização dos vetores
  hipHostMalloc((void **) &a, size, hipHostMallocDefault);
  hipHostMalloc((void **) &b, size, hipHostMallocDefault);
  initvet(a,b);

  // alocação de memória na GPU para os vetores (a,b e c)
  hipMalloc ((void **) &dev_a, size);
  hipMalloc ((void **) &dev_b, size);
  hipMalloc ((void **) &dev_c, size);

  // cópia dos vetores gerados em CPU p/ memória da GPU
  hipMemcpy (dev_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy (dev_b, b, size, hipMemcpyHostToDevice);

  // execução do kernel vecAdd em GPU.
  vecAdd<<<(int)ceil(N/T),T>>>(dev_a, dev_b, dev_c);

  hipHostMalloc((void **) &c, size, hipHostMallocDefault);
  hipMemcpy (c, dev_c, size, hipMemcpyDeviceToHost);
  printvetores (a, b, c);
  printf ("\n **** \n Nro Threads = %d\n Nro Max por Bloco = %d\n Nro de Blocos = %d\n", N, T, (int) ceil(N/T));

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  hipHostFree(a);
  hipHostFree(b);
  hipHostFree(c);

  return 0;
}
