
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>

// CPU: marca o tempo
__host__ double wtime() {
  struct timeval t;
  gettimeofday(&t, NULL);
  return t.tv_sec + (double) t.tv_usec / 1000000;
}

// CPU: Núcleo de execução (processamento)
__host__ void fhcalc(int n)
{
   double v1=0;

    for (int j=0; j < 10000; j++) {
      for(int i=2; i < n; ++i) {
          v1=pow(v1,i);          
       }
     }
}

// GPU: Núcleo de execução (processamento)
__global__ void fdcalc(int n)
{
   double v1=0;

    for (int j=0; j < 10000; j++) {
      for(int i=2; i < n; ++i) {
          v1=pow(v1,i);          
       }
     }
}

// CPU: Função principal
int main (int argc, char ** argv) { 
	int nthreads = 400;
	int nblocos = 1;
	int ncalc = 400;
        double start_time = 0, end_time = 0;

        // Tratamento dos paramêtros
	if (argc == 4) {
		nblocos = atoi(argv[1]);
		nthreads = atoi(argv[2]);
		ncalc = atoi(argv[3]);
	} else {
                printf ("\n ############# \n");
		printf ("./05_proc <nblocos> <nthreads> <nloop>\n");
		printf ("Caso não haja passagem de parâmetros, atribuiu-se:\n(1) nblocos c/ %d, nthreads definido c/ %d e ncalc = %d \n", nblocos, nthreads, ncalc);
	}

        // Mensura o tempo de processamento do kernel em 1 bloco c/ 1 thread
        printf ("\n##### DEVICE (1,1) #####\n");
        start_time = wtime();

        // GPU: Execução do kernel em 1,1
	fdcalc<<<1,1>>>(ncalc);
        
        // CPU: Aguarda a sincronização das threads
	hipDeviceSynchronize();
        end_time = wtime();
        printf("\n");
        printf("\tRuntime: %f\n", end_time - start_time);

       
        // Mensura o tempo de processamento do kernel c/ n blocos e m threads
        printf ("\n##### DEVICE (%d,%d) #####\n", nblocos, nthreads);
        start_time = wtime();

        // GPU: Execução do kernel em n,m
	fdcalc<<<nblocos,nthreads>>>(ncalc/nthreads);

        // CPU: Aguarda a sincronização das threads
	hipDeviceSynchronize();
        end_time = wtime();
        printf("\n");
        printf("\tRuntime: %f\n", end_time - start_time);


        // Mensura o tempo de processamento em CPU
        printf ("\n##### HOST #####\n");
        start_time = wtime();
	fhcalc(ncalc);
	hipDeviceSynchronize();
        end_time = wtime();
        printf("\n");
        printf("\tRuntime: %f\n", end_time - start_time);
	return 0;
}
