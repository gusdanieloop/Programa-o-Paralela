#include <hip/hip_runtime.h>
#include <stdio.h>

#define mega 1048576

__global__ void fdcalc(int n)
{
   long n1 = 0;

    for (int j=0; j < 100000; j++) {
      for(int i=2; i < n; ++i) {
         n1=pow(n1,i);
         //n1=n1*i; (GF 730)
       }
     }
}

// função principal executada iniciada em CPU
int main(int argc, char const *argv[]) {
  unsigned long *dev_a;
  unsigned long *dev_b;
  unsigned long *dev_c;
  unsigned long *dev_d;
  unsigned long *dev_e;
  unsigned long *dev_f;
  int error=0;
  int i=0, mem=0, mem2=0;


  if (argc == 3) {
     mem = atoi(argv[1]);
     mem2 = atoi(argv[2]);
  } else {
     printf ("./06_mem <mem_contigua> <mem_N_contigua>`\n");
     printf ("Titan = ./06_mem 2040 12000\n");
     printf ("730   = ./06_mem 1930 1970\n");
     exit(1);
  }

  printf ("\n #####  ##### \n Alocação de Memória Global\n");
  i = mem;
  while (!error) {
     // alocação de memória na GPU 
     error = hipMalloc ((void **) &dev_a, mega*i);
     printf ("\t hipMalloc (%d MB)\tError Status %d\n", i, error);
     fdcalc <<<1,1>>>(20);
     hipDeviceSynchronize();
     hipFree(dev_a);
     i+=10;
  }

  printf ("\n #####  ##### \n Alocação de Memória Global\n");
  error=0;
  i = (int)mem2/6;
  while (!error) {
     // alocação de memória na GPU 
     error = hipMalloc ((void **) &dev_a, mega*i);
     error = hipMalloc ((void **) &dev_b, mega*i);
     error = hipMalloc ((void **) &dev_c, mega*i);
     error = hipMalloc ((void **) &dev_d, mega*i);
     error = hipMalloc ((void **) &dev_e, mega*i);
     error = hipMalloc ((void **) &dev_f, mega*i);
     printf ("\t hipMalloc (%d MB)\tError Status %d\n", i*6, error);
     fdcalc <<<1,1>>>(20);
     hipDeviceSynchronize();
     hipFree(dev_a);
     hipFree(dev_b);
     hipFree(dev_c);
     hipFree(dev_d);
     hipFree(dev_e);
     hipFree(dev_f);
     i+=10;
  }
  printf("\n");

  return 0;
}
