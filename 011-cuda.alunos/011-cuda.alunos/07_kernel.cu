
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>

// GPU: Impressão dos índices 
__global__ void fIndice() {

   printf ("%d\t%d\t%d\n", threadIdx.x, blockIdx.x, blockDim.x);
}

// CPU: Função principal
int main (int argc, char ** argv) { 
       int nblocos = 0;
       int nthreads = 0;

       // Tratamento dos paramêtros
	if (argc == 3) {
		nblocos = atoi(argv[1]);
		nthreads = atoi(argv[2]);
	} else {
                printf ("\n ############# \n");
		printf ("./07_kernel <nblocos> <nthreads>\n");
		printf ("./07_kernel 32 128  | sort -n | tail -32\n");
		exit(1);
	}

        // Limite bloco 1024, limite de thread 1024
        // Limite MAX (bloco * thread) = 4096 
	fIndice<<<nblocos,nthreads>>>();
	hipDeviceSynchronize();
        printf ("threadIdx.x blockIdx.x blockDim.x\n");

	return 0;
}
