#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

// valores compatíveis devem ter sqrt(N) inteiro (ex.: sqrt(4), sqrt(9), srqt(16), srqt (25)
#define N 4 // Número de linhas de uma matriz quadrada!
            // Número de colunas = número de linhas (matriz quadrada)

// Uma thread por posição da matriz
// Alocação da matriz em bloco contíguo de memória (um vetor N*N)
// Acesso com índices de matrizes (i,j), sendo i=linha e j=coluna
// sqrt(N) é o número de blocos
// sqrt(N) é o número de threads por bloco
// Ex.: Para N=4, tem-se 4 linhas e 4 colunas, totalizando 16 elementos
//      2 blocos c/ 2 threads cada para x (0,1,2,3)
__global__ void matAdd (int *da, int *db, int *dc) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   int j = blockIdx.y * blockDim.y + threadIdx.y;

   printf ("[%d][%d]=%d\t(x)\t%d\t%d\t%d\t(y)\t%d\t%d\t%d\n", \\
           i,j,(i*N+j), threadIdx.x, blockIdx.x, blockDim.x, \\
           threadIdx.y, blockIdx.y, blockDim.y);

   dc[i*N+j] = da[i*N+j] + db[i*N+j];
}

// função executada na GPU
__global__ void vecAdd (int *da, int *db, int *dc) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;

   //printf ("x=(%d)\t%d\t%d\t%d\n", i, threadIdx.x, blockIdx.x, blockDim.x);

   if (i < (N*N)) {
     dc[i] = da[i] + db[i];
   }
}

// função executada na CPU
__host__ void initvet(int *host_a, int *host_b) {
  // Inicialização dos vetores a e b
  for (int i=0; i < N; i++) {
    for (int j=0; j < N; j++) {
       host_b[i*N+j] = (i+j)+((N-1)*i);
       host_a[i*N+j] = (N*N)-host_b[i*N+j];
    }
  }
}

// função executada na CPU
__host__ void printvetores (int *a, int *b, int *c) {
  printf("\t [l,c] \t A\t B\t C\t \n");
  for (int i=0; i < N; i++) {
    for (int j=0; j < N; j++) { 
      // if (((i % 10) == 0) && ((j%10) == 0)) 
      printf("\t [%d,%d] \t %d\t %d\t %d\n", i, j, a[i*N+j], b[i*N+j], c[i*N+j]);
    }
  }
}

// função principal executada iniciada em CPU
int main(int argc, char const *argv[]) {
  int *a, *b, *c;
  int *dev_a, *dev_b, *dev_c;
  int size;

  // Alocação de matriz quadrada
  size = N * N * sizeof(int);

  // Alocação de memória na CPU
  hipHostMalloc((void **) &a, size);
  hipHostMalloc((void **) &b, size);
  hipHostMalloc((void **) &c, size);
  
  // Inicialização dos vetores
  initvet(a,b);

  // Alocação de memória na GPU para os vetores (a,b e c)
  hipMalloc ((void **) &dev_a, size);
  hipMalloc ((void **) &dev_b, size);
  hipMalloc ((void **) &dev_c, size);

  // Cópia dos vetores gerados em CPU p/ memória da GPU
  hipMemcpy (dev_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy (dev_b, b, size, hipMemcpyHostToDevice);

  // Número de linhas/colunas por bloco
  dim3 dimGrid ((int) sqrt(N), (int) sqrt(N)); 
  dim3 dimBlock((int) sqrt(N), (int) sqrt(N)); 

  // GPU: Processamento do kernel matAdd
  //      Uso dos índices como matriz
  printf ("[i][j]=(i*N+j) (x)threadIdx.x blockIdx.x blockDim.x (y)threadIdx.y blockIdx.y blockDim.y\n");
  matAdd<<< dimGrid, dimBlock>>>(dev_a, dev_b, dev_c);
  hipDeviceSynchronize();

  // Cópia do vetor (c) da GPU p/ CPU
  hipMemcpy (c, dev_c, size, hipMemcpyDeviceToHost);

  // Impressão dos vetores
  printf ("\t ##### #####       ##### ##### \n");
  printvetores (a, b, c);
  printf ("\t ##### #####       ##### ##### \n");


  // GPU: Processamento do kernel vecAdd 
  //      Soma c/ índices formato de vetor
  vecAdd<<<N, N>>>(dev_a, dev_b, dev_c);

  // Cópia do vetor (c) da GPU p/ CPU
  hipMemcpy (c, dev_c, size, hipMemcpyDeviceToHost);

  // Impressão dos vetores
  printvetores (a, b, c);
  printf ("\t ##### #####       ##### ##### \n");

  // Liberação de memória GPU
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  // Liberação de memória CPU
  hipHostFree(a);
  hipHostFree(b);
  hipHostFree(c);

  return 0;
}
